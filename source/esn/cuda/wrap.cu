
#include <hip/hip_runtime.h>
__global__ void kernel_srcp(float * v)
{
    v[0] = 1.0f / v[0];
}

__global__ void kernel_stanhv(int n, float * v)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        v[i] = tanh(v[i]);
}

__global__ void kernel_srandv_helper(
    int n, const float * a, const float * b, float * x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        x[i] = x[i] * (*b - *a) + *a;
}

void wrap_srcp(float * v)
{
    kernel_srcp<<<1,1>>>(v);
}

void wrap_stanhv(int n, float * v)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_stanhv<<<gridSize, blockSize>>>(n, v);
}

void wrap_srandv_helper(int n, const float * a, const float * b, float * x)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_srandv_helper<<<gridSize, blockSize>>>(n, a, b, x);
}
