
#include <hip/hip_runtime.h>
__global__ void kernel_sfillv(int n, const float * alpha, float * x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        x[i] = *alpha;
}

__global__ void kernel_srcp(float * v)
{
    v[0] = 1.0f / v[0];
}

__global__ void kernel_stanhv(int n, float * v)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        v[i] = tanh(v[i]);
}

__global__ void kernel_srandv_helper(
    int n, const float * a, const float * b, float * x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        x[i] = x[i] * (*b - *a) + *a;
}

__global__ void kernel_srandspv_helper(
    int n, const float * sparsity, const float * spx, float * x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        if (spx[i] < *sparsity)
            x[i] = 0.0f;
}

__global__ void kernel_sprodvv(int n, const float * x, float * y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] *= x[i];
}

__global__ void kernel_sdivvv(int n, float * x, const float * y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        x[i] /= y[i];
}

void wrap_sfillv(int n, const float * alpha, float * x)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_sfillv<<<gridSize, blockSize>>>(n, alpha, x);
}

void wrap_srcp(float * v)
{
    kernel_srcp<<<1,1>>>(v);
}

void wrap_stanhv(int n, float * v)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_stanhv<<<gridSize, blockSize>>>(n, v);
}

void wrap_srandv_helper(int n, const float * a, const float * b, float * x)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_srandv_helper<<<gridSize, blockSize>>>(n, a, b, x);
}

void wrap_srandspv_helper(int n, const float * sparsity,
    const float * spx, float * x)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_srandspv_helper<<<gridSize, blockSize>>>(n, sparsity, spx, x);
}

void wrap_sprodvv(int n, const float * x, float * y)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_sprodvv<<<gridSize, blockSize>>>(n, x, y);
}

void wrap_sdivvv(int n, float * x, const float * y)
{
    const int blockSize = 128;
    const int gridSize = (n + blockSize - 1) / blockSize;
    kernel_sdivvv<<<gridSize, blockSize>>>(n, x, y);
}
